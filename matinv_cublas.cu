#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
//#include <conio.h>

#define CUDA_CALL(res, str) { if (res != hipSuccess) { printf("CUDA Error : %s : %s %d : ERR %s\n", str, __FILE__, __LINE__, hipGetErrorName(res)); } }
#define CUBLAS_CALL(res, str) { if (res != HIPBLAS_STATUS_SUCCESS) { printf("CUBLAS Error : %s : %s %d : ERR %d\n", str, __FILE__, __LINE__, int(res)); } }

static hipEvent_t cu_TimerStart;
static hipEvent_t cu_TimerStop;

void d_CUDATimerStart(void)
{
    CUDA_CALL(hipEventCreate(&cu_TimerStart), "Failed to create start event!");
    CUDA_CALL(hipEventCreate(&cu_TimerStop), "Failed to create stop event!");

    CUDA_CALL(hipEventRecord(cu_TimerStart), "Failed to record start event!");
}

float d_CUDATimerStop(void)
{
    CUDA_CALL(hipEventRecord(cu_TimerStop), "Failed to record stop event!");

    CUDA_CALL(hipEventSynchronize(cu_TimerStop), "Failed to synch stop event!");

    float ms;

    CUDA_CALL(hipEventElapsedTime(&ms, cu_TimerStart, cu_TimerStop), "Failed to elapse events!");

    CUDA_CALL(hipEventDestroy(cu_TimerStart), "Failed to destroy start event!");
    CUDA_CALL(hipEventDestroy(cu_TimerStop), "Failed to destroy stop event!");

    return ms;
}

float* d_GetInv(float* L, int n)
{
    hipblasHandle_t cu_cublasHandle;
    CUBLAS_CALL(hipblasCreate(&cu_cublasHandle), "Failed to initialize cuBLAS!");

    float** adL;
    float** adC;
    float* dL;
    float* dC;
    int* dLUPivots;
    int* dLUInfo;

    size_t szA = n * n * sizeof(float);

    CUDA_CALL(hipMalloc(&adL, sizeof(float*)), "Failed to allocate adL!");
    CUDA_CALL(hipMalloc(&adC, sizeof(float*)), "Failed to allocate adC!");
    CUDA_CALL(hipMalloc(&dL, szA), "Failed to allocate dL!");
    CUDA_CALL(hipMalloc(&dC, szA), "Failed to allocate dC!");
    CUDA_CALL(hipMalloc(&dLUPivots, n * sizeof(int)), "Failed to allocate dLUPivots!");
    CUDA_CALL(hipMalloc(&dLUInfo, sizeof(int)), "Failed to allocate dLUInfo!");

    CUDA_CALL(hipMemcpy(dL, L, szA, hipMemcpyHostToDevice), "Failed to copy to dL!");
    CUDA_CALL(hipMemcpy(adL, &dL, sizeof(float*), hipMemcpyHostToDevice), "Failed to copy to adL!");
    CUDA_CALL(hipMemcpy(adC, &dC, sizeof(float*), hipMemcpyHostToDevice), "Failed to copy to adC!");

    d_CUDATimerStart();

    CUBLAS_CALL(hipblasSgetrfBatched(cu_cublasHandle, n, adL, n, dLUPivots, dLUInfo, 1), "Failed to perform LU decomp operation!");
    CUDA_CALL(hipDeviceSynchronize(), "Failed to synchronize after kernel call!");

    CUBLAS_CALL(hipblasSgetriBatched(cu_cublasHandle, n, (const float **)adL, n, dLUPivots, adC, n, dLUInfo, 1), "Failed to perform Inverse operation!");
    CUDA_CALL(hipDeviceSynchronize(), "Failed to synchronize after kernel call!");

    float timed = d_CUDATimerStop();

    printf("cublas inverse in: %.5f ms.\n", timed);

    float* res = (float*)malloc(szA);

    CUDA_CALL(hipMemcpy(res, dC, szA, hipMemcpyDeviceToHost), "Failed to copy to res!");

    CUDA_CALL(hipFree(adL), "Failed to free adL!");
    CUDA_CALL(hipFree(adC), "Failed to free adC!");
    CUDA_CALL(hipFree(dL), "Failed to free dL!");
    CUDA_CALL(hipFree(dC), "Failed to free dC!");
    CUDA_CALL(hipFree(dLUPivots), "Failed to free dLUPivots!");
    CUDA_CALL(hipFree(dLUInfo), "Failed to free dLUInfo!");

    CUBLAS_CALL(hipblasDestroy(cu_cublasHandle), "Failed to destroy cuBLAS!");

    return res;
}

int main()
{
    int n = 364;
    float* L = (float*)malloc(n * n * sizeof(float));
    for(int iloop=0; iloop<10; ++iloop) {
        for(int i = 0; i < n * n; i++)
            L[i] = ((float)rand()/(float)(RAND_MAX));
        float* inv = d_GetInv(L, n);
    }

    printf("done.");
    //_getch();

    return 0;
}
