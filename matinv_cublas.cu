#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
//#include <conio.h>

#define CUDA_CALL(res, str) { if (res != hipSuccess) { printf("CUDA Error : %s : %s %d : ERR %s\n", str, __FILE__, __LINE__, hipGetErrorName(res)); } }
#define CUBLAS_CALL(res, str) { if (res != HIPBLAS_STATUS_SUCCESS) { printf("CUBLAS Error : %s : %s %d : ERR %d\n", str, __FILE__, __LINE__, int(res)); } }

static hipEvent_t cu_TimerStart;
static hipEvent_t cu_TimerStop;

void d_CUDATimerStart(void)
{
    CUDA_CALL(hipEventCreate(&cu_TimerStart), "Failed to create start event!");
    CUDA_CALL(hipEventCreate(&cu_TimerStop), "Failed to create stop event!");

    CUDA_CALL(hipEventRecord(cu_TimerStart), "Failed to record start event!");
}

float d_CUDATimerStop(void)
{
    CUDA_CALL(hipEventRecord(cu_TimerStop), "Failed to record stop event!");

    CUDA_CALL(hipEventSynchronize(cu_TimerStop), "Failed to synch stop event!");

    float ms;

    CUDA_CALL(hipEventElapsedTime(&ms, cu_TimerStart, cu_TimerStop), "Failed to elapse events!");

    CUDA_CALL(hipEventDestroy(cu_TimerStart), "Failed to destroy start event!");
    CUDA_CALL(hipEventDestroy(cu_TimerStop), "Failed to destroy stop event!");

    return ms;
}

// L point to the first element of first array
float* d_GetInv(float* L, int n, int b)
{
    hipblasHandle_t cu_cublasHandle;
    CUBLAS_CALL(hipblasCreate(&cu_cublasHandle), "Failed to initialize cuBLAS!");

    float** Lp = (float**)malloc(b * sizeof(float*));
    float** Cp = (float**)malloc(b * sizeof(float*));
    float** adL;
    float** adC;
    float* dL;
    float* dC;
    int* dLUPivots;
    int* dLUInfo;

    size_t szA = n * n * sizeof(float);

    CUDA_CALL(hipMalloc(&adL, b * sizeof(float*)), "Failed to allocate adL!");
    CUDA_CALL(hipMalloc(&adC, b * sizeof(float*)), "Failed to allocate adC!");
    CUDA_CALL(hipMalloc(&dL, b * szA), "Failed to allocate dL!");
    CUDA_CALL(hipMalloc(&dC, b * szA), "Failed to allocate dC!");
    CUDA_CALL(hipMalloc(&dLUPivots, b * n * sizeof(int)), "Failed to allocate dLUPivots!");
    CUDA_CALL(hipMalloc(&dLUInfo, b * sizeof(int)), "Failed to allocate dLUInfo!");

    CUDA_CALL(hipMemcpy(dL, L, b * szA, hipMemcpyHostToDevice), "Failed to copy to dL!");
    // create pointer array for matrices
    for (int i = 0; i < b; i++) {
        Lp[i] = dL + (i*n*n);
    }
    CUDA_CALL(hipMemcpy(adL, Lp, b * sizeof(float*), hipMemcpyHostToDevice), "Failed to copy to adL!");

    for (int i = 0; i < b; i++) {
        Cp[i] = dC + (i*n*n);
    }
    CUDA_CALL(hipMemcpy(adC, Cp, b * sizeof(float*), hipMemcpyHostToDevice), "Failed to copy to adC!");

    d_CUDATimerStart();

    CUBLAS_CALL(hipblasSgetrfBatched(cu_cublasHandle, n, adL, n, dLUPivots, dLUInfo, b), "Failed to perform LU decomp operation!");
    CUDA_CALL(hipDeviceSynchronize(), "Failed to synchronize after kernel call!");

    CUBLAS_CALL(hipblasSgetriBatched(cu_cublasHandle, n, (const float **)adL, n, dLUPivots, adC, n, dLUInfo, b), "Failed to perform Inverse operation!");
    CUDA_CALL(hipDeviceSynchronize(), "Failed to synchronize after kernel call!");

    float timed = d_CUDATimerStop();

    printf("cublas inverse in: %.5f ms.\n", timed);

    float* res = (float*)malloc(b * szA);

    CUDA_CALL(hipMemcpy(res, dC, b * szA, hipMemcpyDeviceToHost), "Failed to copy to res!");

    CUDA_CALL(hipFree(adL), "Failed to free adL!");
    CUDA_CALL(hipFree(adC), "Failed to free adC!");
    CUDA_CALL(hipFree(dL), "Failed to free dL!");
    CUDA_CALL(hipFree(dC), "Failed to free dC!");
    CUDA_CALL(hipFree(dLUPivots), "Failed to free dLUPivots!");
    CUDA_CALL(hipFree(dLUInfo), "Failed to free dLUInfo!");

    CUBLAS_CALL(hipblasDestroy(cu_cublasHandle), "Failed to destroy cuBLAS!");

    free(Lp);
    free(Cp);
    return res;
}

void print(float *m, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%f ", m[i+j*n]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[])
{
    int b = 1;
    int nloop = 100;
    int n = 364;
    if (argc > 1) {
        b = atoi(argv[1]);
    }

    float* L = (float*)malloc(b * n * n * sizeof(float));
    float* inv = 0;
    for(int iloop=0; iloop<nloop; ++iloop) {
        for(int i = 0; i < b * n * n; i++) {
            L[i] = ((float)rand()/(float)(RAND_MAX));
        }
        inv = d_GetInv(L, n, b);
        free(inv);
    }

    //for(int i = 0; i < b * n * n; i++) {
    //    L[i] = 0;
    //}
    //for (int i = 0; i < b; ++i) {
    //    L[0+0*n+i*n*n] = 1;
    //    L[1+1*n+i*n*n] = 10;
    //}
    //inv = d_GetInv(L, n, b);
    //for (int i = 0; i < b; ++i) {
    //    print(L+i*n*n, n);
    //    print(inv+i*n*n, n);
    //    printf("\n");
    //}

    free(L);

    printf("done.");
    //_getch();

    return 0;
}
